#include "hip/hip_runtime.h"
/*
* Copyright 2017-2020 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include "ColorSpace.h"

__constant__ float matYuv2Rgb[3][3];
__constant__ float matRgb2Yuv[3][3];


void inline GetConstants(int iMatrix, float &wr, float &wb, int &black, int &white, int &max) {
    black = 16; white = 235;
    max = 255;

    switch (iMatrix)
    {
    case ColorSpaceStandard_BT709:
    default:
        wr = 0.2126f; wb = 0.0722f;
        break;

    case ColorSpaceStandard_FCC:
        wr = 0.30f; wb = 0.11f;
        break;

    case ColorSpaceStandard_BT470:
    case ColorSpaceStandard_BT601:
        wr = 0.2990f; wb = 0.1140f;
        break;

    case ColorSpaceStandard_SMPTE240M:
        wr = 0.212f; wb = 0.087f;
        break;

    case ColorSpaceStandard_BT2020:
    case ColorSpaceStandard_BT2020C:
        wr = 0.2627f; wb = 0.0593f;
        // 10-bit only
        black = 64 << 6; white = 940 << 6;
        max = (1 << 16) - 1;
        break;
    }
}

void SetMatYuv2Rgb(int iMatrix) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        1.0f, 0.0f, (1.0f - wr) / 0.5f,
        1.0f, -wb * (1.0f - wb) / 0.5f / (1 - wb - wr), -wr * (1 - wr) / 0.5f / (1 - wb - wr),
        1.0f, (1.0f - wb) / 0.5f, 0.0f,
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * max / (white - black) * mat[i][j]);
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(matYuv2Rgb), mat, sizeof(mat));
}

void SetMatRgb2Yuv(int iMatrix) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        wr, 1.0f - wb - wr, wb,
        -0.5f * wr / (1.0f - wb), -0.5f * (1 - wb - wr) / (1.0f - wb), 0.5f,
        0.5f, -0.5f * (1.0f - wb - wr) / (1.0f - wr), -0.5f * wb / (1.0f - wr),
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * (white - black) / max * mat[i][j]);
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(matRgb2Yuv), mat, sizeof(mat));
}

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}

template<class Rgb, class YuvUnit>
__device__ inline Rgb YuvToRgbForPixel(YuvUnit y, YuvUnit u, YuvUnit v) {
    const int 
        low = 1 << (sizeof(YuvUnit) * 8 - 4),
        mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
    const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;
    YuvUnit 
        r = (YuvUnit)Clamp(matYuv2Rgb[0][0] * fy + matYuv2Rgb[0][1] * fu + matYuv2Rgb[0][2] * fv, 0.0f, maxf),
        g = (YuvUnit)Clamp(matYuv2Rgb[1][0] * fy + matYuv2Rgb[1][1] * fu + matYuv2Rgb[1][2] * fv, 0.0f, maxf),
        b = (YuvUnit)Clamp(matYuv2Rgb[2][0] * fy + matYuv2Rgb[2][1] * fu + matYuv2Rgb[2][2] * fv, 0.0f, maxf);
    
    Rgb rgb{};
    const int nShift = abs((int)sizeof(YuvUnit) - (int)sizeof(rgb.c.r)) * 8;
    if (sizeof(YuvUnit) >= sizeof(rgb.c.r)) {
        rgb.c.r = r >> nShift;
        rgb.c.g = g >> nShift;
        rgb.c.b = b >> nShift;
    } else {
        rgb.c.r = r << nShift;
        rgb.c.g = g << nShift;
        rgb.c.b = b << nShift;
    }
    return rgb;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void YuvToRgbKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgb, int nRgbPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    *(RgbIntx2 *)pDst = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y).d,
        YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y).d,
    };
    *(RgbIntx2 *)(pDst + nRgbPitch) = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y).d, 
        YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y).d,
    };
}

template <class COLOR32>
void Nv12ToColor32(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

// Explicit Instantiation
template void Nv12ToColor32<BGRA32>(uint8_t* dpNv12, int nNv12Pitch, uint8_t* dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
